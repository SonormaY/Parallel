#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>


__global__ void generateMatrixCUDA(int *matrix, int n, unsigned long seed) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        hiprandState state;
        hiprand_init(seed, row * n + col, 0, &state);
        matrix[row * n + col] = hiprand(&state) % 100;
    }
}

__global__ void multiplyMatricesCUDA(int *matA, int *matB, int *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += matA[row * n + i] * matB[i * n + col];
        }
        result[row * n + col] = sum;
    }
}

void printMatrix(const int* matrix, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << matrix[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int size;
    std::cout << "Enter the size of the square matrix: ";
    std::cin >> size;

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size * size * sizeof(int));
    hipMalloc(&d_B, size * size * sizeof(int));
    hipMalloc(&d_C, size * size * sizeof(int));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x, (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    unsigned long seed = time(0);
    generateMatrixCUDA<<<numBlocks, threadsPerBlock>>>(d_A, size, seed);
    generateMatrixCUDA<<<numBlocks, threadsPerBlock>>>(d_B, size, seed + 1);
    hipDeviceSynchronize();

    int *h_A = new int[size * size];
    int *h_B = new int[size * size];
    hipMemcpy(h_A, d_A, size * size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, size * size * sizeof(int), hipMemcpyDeviceToHost);

    if (size < 10)
    {
        std::cout << "Matrix A:" << std::endl;
        printMatrix(h_A, size);
        std::cout << "Matrix B:" << std::endl;
        printMatrix(h_B, size);
    }
    
    

    auto start = std::chrono::high_resolution_clock::now();
    multiplyMatricesCUDA<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, size);
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = stop - start;

    int *h_C = new int[size * size];
    hipMemcpy(h_C, d_C, size * size * sizeof(int), hipMemcpyDeviceToHost);

    if (size < 10)
    {
        std::cout << "Part of Result Matrix:" << std::endl;
        printMatrix(h_C, size);
    }
    

    std::cout << "Time taken for matrix multiplication (CUDA): " << duration.count() * 1000 << " ms" << std::endl;

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
